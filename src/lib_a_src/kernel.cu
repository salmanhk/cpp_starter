#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void addKernel(int* c, const int* a, const int* b, int size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < size) {
    c[i] = a[i] + b[i];
  }
}

extern "C" void addWithCuda(int* c, const int* a, const int* b, int size) {
  int* dev_a = nullptr;
  int* dev_b = nullptr;
  int* dev_c = nullptr;

  // Allocate GPU buffers for three vectors (two input, one output).
  hipMalloc((void**)&dev_c, size * sizeof(int));
  hipMalloc((void**)&dev_a, size * sizeof(int));
  hipMalloc((void**)&dev_b, size * sizeof(int));

  // Copy input vectors from host memory to GPU buffers.
  hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

  // Launch a kernel on the GPU.
  int threadsPerBlock = 256;
  int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
  addKernel << <blocksPerGrid, threadsPerBlock >> > (dev_c, dev_a, dev_b, size);

  // hipDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  hipDeviceSynchronize();

  // Copy output vector from GPU buffer to host memory.
  hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(dev_c);
  hipFree(dev_a);
  hipFree(dev_b);
}
